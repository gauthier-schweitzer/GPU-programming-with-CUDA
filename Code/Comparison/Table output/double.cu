#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/**************************************************************
The code in time.h is a part of a course on cuda taught by its authors:
Lokman A. Abbas-Turki
**************************************************************/
#include "timer.h"

/**************************************************************
Common functions
**************************************************************/

// Compare function for qsort
int compare_function(const void *a,const void *b) {
    double *x = (double *) a;
		double *y = (double *) b;
    if (*x < *y) return - 1;
    else if (*x > *y) return 1;
    return 0;
}


// Generate gaussian vector using Box Muller
void gaussian_vector(double *v, double mu, double sigma, int n) {

    for (int i = 0; i<n; i++){
		    double u1 = (double)rand()/(double)(RAND_MAX);
		    double u2 = (double)rand()/(double)(RAND_MAX);
		    v[i] = sigma * (sqrt( -2 * log(u1)) * cos(2 * M_PI * u2)) + mu;
	  }
}


//Function to print a small vector of doubles on host
void print_vector(double *c, int m, int n) {

    for (int i=0; i<m; i++){
        printf("%f     ", c[i]);
        printf("\n");
 	  }
}


// Kernel for computing the square of a vector (INPLACE)
// We actually only need z ** 2 in the computations and not z
// The square norm is also computed
void square_vector(double *z, double *znorm, int n){
		for (int i = 0; i < n; i++) {
				double zi = z[i];
				double zsqri = zi * zi;
				z[i] = zsqri;
				znorm[0] += zsqri;
		}
}


// Function for computing f (the secular function of interest) at a given point x
double secfunc(double *d, double *zsqr, double rho, double x, int n) {

    double sum = 0;
    for (int i=0; i < n; i++){
        sum += zsqr[i] / (d[i] - x);
	  }

    return rho + sum;
}


// Function for computing f' (the prime derivative of the secular function of interest) at a given point x
double secfunc_prime(double *d, double *zsqr, double x, int n) {

    double sum = 0;
    for (int i=0; i < n; i++){
        int di = d[i];
		    sum += zsqr[i] / ((di - x) * (di - x));
    }

	  return sum;
}


// Device function for computing f'' (the second derivative of the secular function of interest)
double secfunc_second(double *d, double *zsqr, double x, int n){
    double sum = 0;

		for (int i = 0; i < n; i++) {
		    double di = d[i];
				sum += zsqr[i] / ((di - x) * (di - x) * (di - x));
		}

		return 2 * sum;
}


// Useful intermediary function, see equations (30) and (31) from Li's paper on page 13 and equation (42) on page 20
double discrimant_int(double a, double b, double c){

    if (a <= 0) return (a - sqrtf(a * a - 4 * b * c)) / (2 * c);
    else return (2 * b) / (a + sqrtf(a * a - 4 * b *c));
}


// Useful intermediary function, see equation (46) from Li's paper on page 21
double discrimant_ext(double a, double b, double c){

    if (a >= 0) return (a + sqrtf(a * a - 4 * b * c)) / (2 * c);
    else return (2 * b) / (a - sqrtf(a * a - 4 * b *c));
}


// h partition of the secular function, used for Initialization
double h_secfunc(double d_k, double d_kplus1, double zsqr_k, double zsqr_kplus1, double x){

    return zsqr_k / (d_k - x) + zsqr_kplus1 / (d_kplus1 - x);
}


// Initialization for interior roots (see section 4 of Li's paper - initial guesses from page 18)
double initialization_int(double *d, double *zsqr, double rho, int k, int n){

    double d_k = d[k];
    double d_kplus1 = d[k + 1];
    double zsqr_k = zsqr[k];
    double zsqr_kplus1 = zsqr[k + 1];
    double middle = (d_k + d_kplus1) / 2;
    double delta = d_kplus1 - d_k;
    double f = secfunc(d, zsqr, rho, middle, n);
    double c = f - h_secfunc(d_k, d_kplus1, zsqr_k, zsqr_kplus1, middle);

    if (f >= 0){
        double a = c * delta + zsqr_k + zsqr_kplus1;
        double b = zsqr_k * delta;
        return discrimant_int(a, b, c) + d_k;
    }

    else {
        double a = - c * delta + zsqr_k + zsqr_kplus1;
        double b = - zsqr_kplus1 * delta;
        return discrimant_int(a, b, c) + d_kplus1;
    }
}


// Initialization for the exterior root (see section 4 of Li's paper - initial guesses from page 18)
double initialization_ext(double *d, double *zsqr, double *znorm, double rho, int n){

    double d_nminus1 = d[n - 1];
    double d_nminus2 = d[n - 2];
    double d_n = d_nminus1 + znorm[0] / rho;
    double zsqr_nminus1 = zsqr[n - 1];
    double zsqr_nminus2 = zsqr[n - 2];
    double middle = (d_nminus1 + d_n) / 2;
    double f = secfunc(d, zsqr, rho, middle, n);
    if (f <= 0){
        double hd = h_secfunc(d_nminus2, d_nminus1, zsqr_nminus2, zsqr_nminus1, d_n);
        double c = f - h_secfunc(d_nminus2, d_nminus1, zsqr_nminus2, zsqr_nminus1, middle);
        if (c <= - hd) {
            return d_n;
        }

        else {
            double delta = d_nminus1 - d_nminus2;
            double a = - c * delta + zsqr_nminus2 + zsqr_nminus1;
            double b = - zsqr_nminus1 * delta;
            return discrimant_ext(a, b, c) + d_n;
        }
    }

    else {
        double delta = d_nminus1 - d_nminus2;
        double c = f - h_secfunc(d_nminus2, d_nminus1, zsqr_nminus2, zsqr_nminus1, middle);
        double a = - c * delta + zsqr_nminus2 + zsqr_nminus1;
        double b = - zsqr_nminus1 * delta;
        return discrimant_ext(a, b, c) + d_n;
    }
}


// Computation of a from the paper (page 13)
double a_gragg(double f, double fprime, double delta_k, double delta_kplus1){

    return (delta_k + delta_kplus1) * f - delta_k * delta_kplus1 * fprime;

}


// Computation of b from the paper (page 13)
double b_gragg(double f, double delta_k, double delta_kplus1){

    return delta_k * delta_kplus1 * f;
}


// Computation of c from the section Gragg of the paper (page 15)
double c_gragg(double f, double fprime, double fsecond, double delta_k, double delta_kplus1){

    return f - (delta_k + delta_kplus1) * fprime + delta_k * delta_kplus1 * fsecond / 2.0;

}


// Compute of the update for x (eta) for the interior roots (see section 3.1 - Iteration fomulas, pages 12 and 13)
double eta_int(double d_k, double d_kplus1, double f, double fprime, double fsecond, double x, int k, int n){

    double delta_k = d_k - x;
    double delta_kplus1 = d_kplus1 - x;
    double a = a_gragg(f, fprime, delta_k, delta_kplus1);
    double b = b_gragg(f, delta_k, delta_kplus1);
    double c = c_gragg(f, fprime, fsecond, delta_k, delta_kplus1);
    double eta = discrimant_int(a, b, c);
    return eta;
}

// Compute of the update of x (+eta) for the exterior root
double eta_ext(double d_nminus2, double d_nminus1, double f, double fprime, double fsecond, double x, int n){

    double delta_nminus2 = d_nminus2 - x;
    double delta_nminus1 = d_nminus1 - x;
    double a = a_gragg(f, fprime, delta_nminus2, delta_nminus1);
    double b = b_gragg(f, delta_nminus2, delta_nminus1);
    double c = c_gragg(f, fprime, fsecond, delta_nminus2, delta_nminus1);
    double eta = discrimant_ext(a, b, c);
    return eta;
}

// Iterate to find the k-th interior root
double find_root_int(double *d, double *zsqr, double rho, double x, int k, int n, int maxit, double epsilon, double*loss_CPU){

    int i = 0;
    double f = secfunc(d, zsqr, rho, x, n);;
    double d_k = d[k];
    double d_kplus1 = d[k + 1];

    while ((i < maxit) && (fabsf(f) > epsilon)){
        f = secfunc(d, zsqr, rho, x, n);
        double fprime = secfunc_prime(d, zsqr, x, n);
        double fsecond = secfunc_second(d, zsqr, x, n);
        double eta = eta_int(d_k, d_kplus1, f, fprime, fsecond, x, k, n);
        x += eta;
        i ++;
    }
    *loss_CPU += (double)(abs(f)/n);

    return x;
}


// Iterate to  find the last root (the exterior one)
double find_root_ext(double *d, double *zsqr, double rho, double x, int n, int maxit, double epsilon, double*loss_CPU){

    int i = 0;
    double d_nminus2 = d[n - 2];
    double d_nminus1 = d[n - 1];
    double f = secfunc(d, zsqr, rho, x, n);

    while ((i < maxit) && (fabsf(f) > epsilon)){
        f = secfunc(d, zsqr, rho, x, n);
        double fprime = secfunc_prime(d, zsqr, x, n);
        double fsecond = secfunc_second(d, zsqr, x, n);
        double eta = eta_ext(d_nminus2, d_nminus1, f, fprime, fsecond, x, n);
        x += eta;
        i ++;
    }
    *loss_CPU += (float)(abs(f)/n);
    return x;
}


void find_roots(double *xstar, double *x0, double *d, double *zsqr, double *znorm, double rho, int n, int maxit, float epsilon, double *loss_CPU){
    // We make sure that the loss is set to 0
    *loss_CPU =0;
		for (int i=0; i<n-1; i++){
				xstar[i] = find_root_int(d, zsqr, rho, x0[i], i, n, maxit, epsilon, loss_CPU);
		}

		xstar[n - 1] = find_root_ext(d, zsqr, rho, x0[n - 1], n, maxit, epsilon, loss_CPU);
}


void initialize_x0(double *x0, double *d, double *zsqr, double *znorm, double rho, int n){

		for (int i=0; i<n-1; i++){
				x0[i] = initialization_int(d, zsqr, rho, i, n);
		}

		x0[n - 1] = initialization_ext(d, zsqr, znorm, rho,  n);
}



int main (void) {
    /****************** Access for writing ******************/
    FILE *f = fopen("result_double.csv", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }
    fprintf(f, "n;iter;niter;time_GPU;time_CPU_double;loss_GPU;loss_CPU_double\n");

    /****************** Declaration ******************/
    // Declare vectors or floats
    double *d, *z, *zsqr, *znorm, *x0, *xstar, *loss_GPU, *loss_CPU;


    // rho parameter
    double rho = 10;


    // Size of arrow matrix chosen by the user
    int n, nlow, nhigh, step, niter;
    printf("\nLowest n to test? \n");
    scanf("%d", &nlow);
    printf("\nHighest n to test? \n");
    scanf("%d", &nhigh);
    printf("\nSize of the step? \n");
    scanf("%d", &step);
    printf("\nNumber of iterations of the same n to avoid stochastic error? \n");
    scanf("%d", &niter);
    printf("\n \n******************* CHOICE OF N ******************** \n");
    printf("We compare the chosen algorithms every %d n, for n between %d and %d \n", step, nlow, nhigh);
    printf("Each test is repeated %d times \n\n", niter);
    printf("\n \n********************** TESTS *********************** \n");


    for(n=nlow; n<=nhigh; n+=step){

      //Maximum number of iterations
      int maxit = 1e4;

      //Stopping criterion
      double epsilon = 1e-6;

      // Memory allocation for data
      d = (double*)malloc(n*sizeof(double));
      z = (double*)malloc(n*sizeof(double));
      zsqr = (double*)malloc(n*sizeof(double));

      for (int iter =0; iter<niter; iter++){
        // Memory allocation for computation
    		znorm = (double*)malloc(sizeof(double));
    		x0 = (double*)malloc(n*sizeof(double));
        xstar = (double*)malloc(n*sizeof(double));
        loss_GPU = (double*)malloc(sizeof(double));
        loss_CPU = (double*)malloc(sizeof(double));

        // Create instance of class Timer
        Timer TimG, TimC;


        //Fill the vector d with linear function of n
        for (int i=0; i < n; i++){
            d[i] = 2 * n - i;
        }

        // sort the vector in ascending order
        qsort(d, n, sizeof(double), compare_function);

        // Gaussian rank 1 perturbation
        float mu_z = 5;
        float sigma_z = 1;
        gaussian_vector(z, mu_z, sigma_z, n);
        gaussian_vector(zsqr, mu_z, sigma_z, n);


        /*************************************************************************
        ********************************* CPU ************************************
        *************************************************************************/
        // Start timer CPU
        TimC.start();

        // We first compute the square and squared norm
        square_vector(zsqr, znorm, n);

        // Initialization of x0
        initialize_x0(x0, d, zsqr, znorm, rho, n);


        /***************** Root computation ****************/
        // Find roots
        find_roots(xstar, x0, d, zsqr, znorm, rho, n, maxit, epsilon, loss_CPU);

        // End timer
        TimC.add();

        // Record the performance
        *loss_GPU =0;
        fprintf(f, "%d;%d;%d;%f;%f;%f;%f\n", n, iter, niter, (double)TimG.getsum(), (float)TimC.getsum(), *loss_GPU, *loss_CPU);

        // Free memory used for computation on CPU
        free(znorm);
        free(xstar);
        free(loss_CPU);
        free(loss_GPU);
      }

    printf("%d has been tested\n", n);
    // Free memory used to store data on CPU
    free(d);
    free(z);
    free(zsqr);
    }
    printf("\n \n");

    // We close the access to the file
    fclose(f);
}
